#include "gpu_operators.cuh"

namespace gpu_sql {

hipError_t MemoryManager::allocateDeviceMemory(void** ptr, size_t size) {
    return hipMalloc(ptr, size);
}

hipError_t MemoryManager::freeDeviceMemory(void* ptr) {
    return hipFree(ptr);
}

hipError_t MemoryManager::copyHostToDevice(
    void* dst,
    const void* src,
    size_t size,
    hipStream_t stream
) {
    return hipMemcpyAsync(dst, src, size, hipMemcpyHostToDevice, stream);
}

hipError_t MemoryManager::copyDeviceToHost(
    void* dst,
    const void* src,
    size_t size,
    hipStream_t stream
) {
    return hipMemcpyAsync(dst, src, size, hipMemcpyDeviceToHost, stream);
}

} // namespace gpu_sql
